#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<fstream>
#include<iostream>
#include<sstream>
#include<hip/hip_runtime.h>
#include<cstdlib>
#include<string>

float a[3000][3000], h[10][10],c[3000][3000];
using namespace std;

//kernel function to run on a single thread
__global__ void conv2D(float *d_c, float *d_a, float *d_h,int arows,int acols,int hrows,int hcols)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int i, j;
	float sum;

	__shared__ float  shared_h[100];

	if (idx % 16 < hcols && idy % 16 < hrows) {
		int t = (idy % 16)*hcols + (idx % 16);
		shared_h[t] = d_h[t];
	}

	if (idy < (arows + hrows - 1) && idx < (acols + hcols - 1)) {
		sum = 0;
		for (i = 0; i < hrows; i++) {
			for (j = 0; j < hcols; j++) {
				if (!((idy - i) < 0 || (idx - j) < 0 || (idy - i) >= arows || (idx - j) >= acols)) {
					sum += d_a[((idy - i)*acols) + (idx - j)] * shared_h[i*hcols + j];
				}
			}
		}
		d_c[idy*(acols + hcols - 1) + idx] = sum;
	}
}

int main(int argc, char** argv)
{
	hipError_t err = hipSuccess;
	int acolst, arowst, hrowst, hcolst, a_elems, h_elems;
	string line;
	char* input_file;
	float test;
	int i, j;


	//read from file to get size of a and h
	input_file = argv[1];//"A:/i1024.txt";

	a_elems = 0;
	acolst = 0;
	arowst = 0;
	h_elems = 0;
	hcolst = 0;
	hrowst = 0;

	//read from file to get size of a and h
	ifstream file(input_file);
	if (file.is_open()) {
		i = 0;
		while (getline(file, line) && line != "") {
			j = 0;
			stringstream ss(line);
			while (ss >> test) {
				a[i][j] = test;
				j++;
				a_elems++;
			}
			i++;
			arowst++;
		}


		i = 0;
		while (getline(file, line) && line != "") {
			j = 0;
			stringstream ss(line);
			while (ss >> test) {
				h[i][j] = test;
				j++;
				h_elems++;
			}
			i++;
			hrowst++;
		}

	}
	file.close();

	acolst = a_elems / arowst;
	hcolst = h_elems / hrowst;

	const int acols = acolst;
	const int arows = arowst;
	const int hrows = hrowst;
	const int hcols = hcolst;

	//cout << acols << " " << arows << " " << hcols << " " << hrows << endl;
	
	//assign sizes to a,h,c
	float* h_a = new float[arows*acols];

	float* h_h = new float[hrows*hcols];

	float* h_c = new float[(arows+hrows-1)*(acols+hcols-1)];

	for (i = 0; i < arows; i++) {
		for (j = 0; j < acols; j++) {
			h_a[(i*acols) + j] = a[i][j];
			//cout << h_a[(i*acols)+j]<< " ";
		}
		//cout << endl;
	}
	//cout << endl;

	for (i = 0; i < hrows; i++) {
		for (j = 0; j < hcols; j++) {
			h_h[(i*hcols)+j] = h[i][j];
			//cout << h_h[(i*hcols)+j] << " ";

		}
		//cout << endl;
	}
	//cout << endl;

	size_t size_a = arows*acols * sizeof(float);
	size_t size_h = hrows*hcols * sizeof(float);
	size_t size_c = (arows + hrows - 1)*(acols + hcols - 1) * sizeof(float);

	cout << "So the file is being read" << endl;;
	//inputs and outputs on the host
	//done as global 

	//declare GPU memory pointers
	float *d_a, *d_h, *d_c;
	
	
	//allocate GPU memory
	err = hipMalloc((void**)&d_a, size_a);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector a (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err=hipMalloc((void**)&d_h, size_h);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector h (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err=hipMalloc((void**)&d_c, size_c);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector c (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	
	
	
	//transfer the data to GPU
	err=hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy h_a to d_a (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_h, h_h, size_h, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy h_h to d_h (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	dim3 num_threadblocks(((arows + hrows - 2) / 16) + 1, ((acols + hcols - 2) / 16) + 1, 1);
	/*int x = (arows + hrows - 2) / 16) + 1;
	int y = (acols + hcols - 2) / 16) + 1;
	cout << "no of blocks launched" << x << "x" << y << endl;
	*/
	dim3 threads_per_block(16, 16, 1);


	cout << "Launching kernel";

	//launch kernel on GPU_
	conv2D <<<num_threadblocks+num_threadblocks%threads_per_block, threads_per_block >>> (d_c,d_a,d_h,arows,acols,hrows,hcols);
	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch conv2D kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	err = hipDeviceSynchronize();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}




	//transfer result to CPU
	hipMemcpy(h_c, d_c, size_c, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy d_c to h_c (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}



	cout << "And it's done... Data is out of GPU and in CPU memory"<<endl;


/*
	//Calc conv2D on CPU
	kCenterX = hcols / 2;
	kCenterY = hrows / 2;

	for (i = 0; i < arows; ++i){
		for (j = 0; j < acols; ++j){
			for (m = 0; m < hrows; ++m){     // kernel rows
				mm = hrows - 1 - m;          // row index of flipped kernel
				for (n = 0; n < hcols; ++n){ // kernel columns
					nn = hcols - 1 - n;      // column index of flipped kernel		 
					ii = i + (m - kCenterY);
					jj = j + (n - kCenterX);// index of input signal, used for checking boundary

					// ignore input samples which are out of bound
					if (ii >= 0 && ii < arows && jj >= 0 && jj < acols)
						h_c[i][j] += h_a[ii][jj] * h_h[mm][nn];

				}
			}
		}
	}*/

	//print output
	for (i = 0; i < arows+hrows-1; i++) {
		for (j = 0; j < acols+hcols-1; j++) {
			//h_h[i][j] = h[i][j];
			cout << h_c[(i*(acols+hcols-1))+j] << " ";
		}
		cout << endl;
	}
	cout << endl;
	
	//free GPU memory location
	err = hipFree(d_a);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err=hipFree(d_h);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err=hipFree(d_c);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	////Driver to reset all state
	//err = hipDeviceReset();

	//if (err != hipSuccess)
	//{
	//	fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
	//	exit(EXIT_FAILURE);
	//}
	
	return(0);
}

